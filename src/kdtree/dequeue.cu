#include "hip/hip_runtime.h"
#pragma once
#include "dequeue.h"

/*
////////////////////////////////////////////////////////////////////////
--------------------Priority Queue Implementation-----------------------
////////////////////////////////////////////////////////////////////////
*/

__host__ __device__ int int_log2(int x) {
	int ret = 0;
	while (x >>= 1) ++ret;
	return ret;
}

__host__ __device__ void PriorityQueue::insert(Cuda_KdNode* node, float weight) {
	//assert(size < MAX_Q); // Must not overfill array
	//Q_Node newN = { node, weight };

	// If full, replace the current highest with this, only if it is lower
	if (size == MAX_Q) {
		if (peekMax().priority >= weight) {
			removeMax();
		}
		else {
			return;
		}
	}
	size += 1;
	int idx = size - 1;
	data[idx] = { node, weight };
	bubbleUp(idx);
	//while (idx > 0) {
	//	int parent_idx = parent(idx);
	//	if (data[parent_idx].priority > weight) {
	//		data[idx] = data[parent_idx];
	//		idx = parent_idx;
	//	}
	//	else {
	//		break;
	//	}
	//}
	//data[idx] = { node, weight };
}

__host__ __device__ void PriorityQueue::swap(int idx1, int idx2) {
	Q_Node original_idx1_val = data[idx1];
	data[idx1] = data[idx2];
	data[idx2] = original_idx1_val;
}

__host__ __device__ void PriorityQueue::bubbleUp(int idx) {
	if (int_log2(idx + 1) % 2 == 0) {
		if (parent(idx) >= 0 && data[idx].priority > data[parent(idx)].priority) {
			swap(idx, parent(idx));
			bubbleUpMax(parent(idx));
		}
		else {
			bubbleUpMin(idx);
		}
	}
	else {
		if (parent(idx) >= 0 && data[idx].priority < data[parent(idx)].priority) {
			swap(idx, parent(idx));
			bubbleUpMin(parent(idx));
		}
		else {
			bubbleUpMax(idx);
		}
	}
}

__host__ __device__ void PriorityQueue::bubbleUpMin(int idx) {
	// While idx has a grandparent
	while (idx >= 3) {
		int grandparent_idx = parent(parent(idx));
		if (data[idx].priority < data[grandparent_idx].priority) {
			swap(idx, grandparent_idx);
			idx = grandparent_idx;
		}
		else {
			break;
		}
	}
}

__host__ __device__ void PriorityQueue::bubbleUpMax(int idx) {
	// While idx has a grandparent
	while (idx >= 3) {
		int grandparent_idx = parent(parent(idx));
		if (data[idx].priority > data[grandparent_idx].priority) {
			swap(idx, grandparent_idx);
			idx = grandparent_idx;
		}
		else {
			break;
		}
	}
}

__host__ __device__ Q_Node PriorityQueue::removeMin() {
	//assert(size > 0); // Canot remove from empty queue
	Q_Node to_return = data[0];
	size--;
	data[0] = data[size];
	percolateDown(data[size], 0);
	return to_return;
}

__host__ __device__ int PriorityQueue::largest_idx() {
	int largest;
	if (size >= 2) {
		if (size == 2) {
			// Only 2 nodes, so maximum is the only node on level 1 (the first max level)
			largest = 1;
		}
		else {
			// Largest of left or right on level 1
			if (data[1].priority > data[2].priority) {
				largest = 1;
			}
			else { largest = 2; }
		}
	}
	else { largest = 0; } // Only one node
	return largest;
}

__host__ __device__ Q_Node PriorityQueue::removeMax() {
	int largest = largest_idx();
	Q_Node to_return = data[largest];
	size--;
	data[largest] = data[size];
	percolateDown(data[size], largest);
	return to_return;
}

__host__ __device__ Q_Node PriorityQueue::peekMax() {
	int largest = largest_idx();
	return data[largest];
}

__host__ __device__ void PriorityQueue::percolateDown(Q_Node item, int idx) {
	if (int_log2(idx + 1) % 2 == 0) {
		// Even, min-level
		percolateDownMin(item, idx);
	}
	else {
		// Odd, max-level
		percolateDownMax(item, idx);
	}
	//// If smallest child is less than item
	//if (data[smallestChild].priority < item.priority) {
	//	// Move smallest child up
	//	data[idx] = data[smallestChild];
	//	idx = smallestChild;
	//}
	//// Otherwise done
	//else {
	//	break;
	//}
}

__host__ __device__ void PriorityQueue::percolateDownMin(Q_Node item, int idx) {
	// While node at idx has children
	while (childLeft(idx) < size) {
		// Find the smallest among children and grandchildren
		int descendents[] = {/*childLeft(idx), */childRight(idx),
			childLeft(childLeft(idx)), childRight(childLeft(idx)),
			childLeft(childRight(idx)), childRight(childRight(idx)) };
		int smallestChild = childLeft(idx);
		for (int desc_idx : descendents) {
			if (desc_idx < size && data[desc_idx].priority < data[smallestChild].priority) {
				smallestChild = desc_idx;
			}
		}

		if (smallestChild > childRight(idx)) {
			// smallest is a grandchild
			if (data[smallestChild].priority < data[idx].priority) {
				swap(idx, smallestChild);
				if (data[smallestChild].priority > data[parent(smallestChild)].priority) {
					swap(parent(smallestChild), smallestChild);
				}
				idx = smallestChild;
			}
			else {
				break;
			}
		}
		else {
			// smallest is a child
			if (data[smallestChild].priority < data[idx].priority) {
				swap(idx, smallestChild);
				idx = smallestChild;
			}
			break;
		}
	}
}
__host__ __device__ void PriorityQueue::percolateDownMax(Q_Node item, int idx) {
	// While node at idx has children
	while (childLeft(idx) < size) {
		// Find the smallest among children and grandchildren
		int descendents[] = {/*childLeft(idx), */childRight(idx),
			childLeft(childLeft(idx)), childRight(childLeft(idx)),
			childLeft(childRight(idx)), childRight(childRight(idx)) };
		int smallestChild = childLeft(idx);
		for (int desc_idx : descendents) {
			if (desc_idx < size && data[desc_idx].priority > data[smallestChild].priority) {
				smallestChild = desc_idx;
			}
		}

		if (smallestChild > childRight(idx)) {
			// smallest is a grandchild
			if (data[smallestChild].priority > data[idx].priority) {
				swap(idx, smallestChild);
				if (data[smallestChild].priority < data[parent(smallestChild)].priority) {
					swap(parent(smallestChild), smallestChild);
				}
				idx = smallestChild;
			}
			else {
				break;
			}
		}
		else {
			// smallest is a child
			if (data[smallestChild].priority > data[idx].priority) {
				swap(idx, smallestChild);
				idx = smallestChild;
			}
			break;
		}
	}
}