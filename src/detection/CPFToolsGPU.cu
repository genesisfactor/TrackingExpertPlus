#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <cuda/cutil_math.h>
#include <cuda/cutil_matrix.h>
#include "CPFToolsGPU.h"

#define M_PI 3.14159265359

namespace nsCPFToolsGPU {
	int angle_bins = 12;

	float3* vectorsA;
	float3* vectorsB;
	float3* pcN;
	Matches* pt_matches;
	int* int_p;

	//GetReferenceFrames
	float4* RefFrames;

	//DiscretizeCurvature
	float2* curvature_pairs;
	int* discretized_curvatures;

	//DiscretizeCPF
	CPFDiscreet discretized_cpfs;
}

using namespace texpert;
using namespace nsCPFToolsGPU;

void vecToPointer3F(float3* dst, const vector<Eigen::Vector3f>& src)
{
	Eigen::Vector3f curVec;
	for (int i = 0; i < src.size(); i++) {
		curVec = src.at(i);
		dst[i] = make_float3(curVec(0), curVec(1), curVec(2));
	}
}

void pointerToVecM4F(vector<Eigen::Affine3f>& dst, float4* src)
{
	Eigen::Matrix4f curMatrix;
	for (int i = 0; i < dst.size(); i++)
	{
		curMatrix <<
			(src[i * 4].x, src[i * 4].y, src[i * 4].z, src[i * 4].w,
				src[(i * 4) + 1].x, src[(i * 4) + 1].y, src[(i * 4) + 1].z, src[(i * 4) + 1].w,
				src[(i * 4) + 2].x, src[(i * 4) + 2].y, src[(i * 4) + 2].z, src[(i * 4) + 2].w,
				src[(i * 4) + 3].x, src[(i * 4) + 3].y, src[(i * 4) + 3].z, src[(i * 4) + 3].w);

		dst.at(i) = curMatrix;
	}
}

void pointerToVecI(std::vector<uint32_t>& dst, int* src)
{
	for (int i = 0; i < dst.size(); i++)
		dst.at(i) = src[i];
}



//static
__host__ __device__
float AngleBetweenGPU(const float3 a, const float3 b)
{
	float3 a_norm = normalize(a);
	float3 b_norm = normalize(b);
	float3 c = cross(a, b);
	return atan2f(sqrtf(powf(c.x, 2) + powf(c.y, 2) + powf(c.z, 2)), dot(a_norm, b_norm));
}

float CPFToolsGPU::AngleBetween(const Eigen::Vector3f& a, const Eigen::Vector3f& b)
{
	return AngleBetweenGPU(make_float3(a(0), a(1), a(2)), make_float3(b(0), b(1), b(2)));
}



__global__
void GetRefFrameGPU(float3* p, float3* n, int numPts, float4* res)
{
	int i = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (i >= numPts) return;

	float3 point = p[i] * -1; //the point is never used in its original form, only in its negated form
	float3 normal = n[i];

	float3 axis = cross(normal, make_float3(1, 0, 0));

	if (axis.x == 0.0f && axis.y == 0.0f && axis.z == 0.0f)
	{
		axis = make_float3(1, 0, 0);
	}
	else {
		axis = normalize(axis);
	}

	// create an angle axis transformation that rotates A degrees around the axis.
	float theta =	AngleBetweenGPU(normal, make_float3(1, 0, 0)); //Angle between the surface normal and the x axis

	float cost =	cosf(theta); // cos(theta)
	float omcost =	1 - cost; // 1 - cos(theta)
	float sint =	sinf(theta); // sin(theta)

	float xy =		axis.x * axis.y; // ux * uy
	float yz =		axis.y * axis.z; // uy * uz
	float xz =		axis.x * axis.z; // ux * uz


	float3* rot = (float3*)malloc(3 * sizeof(float3));
	rot[0] = make_float3(cost + (powf(axis.x, 2) * omcost), (xy * omcost) - (axis.z * sint), (xz * omcost) + (axis.y * sint));
	rot[1] = make_float3((xy * omcost) + (axis.z * sint), cost + (powf(axis.y, 2) * omcost), (yz * omcost) - (axis.x * sint));
	rot[2] = make_float3((xz * omcost) - (axis.y * sint), (yz * omcost) + (axis.x * sint), cost + (powf(axis.z, 2) * omcost));

	// create the reference frame
	res[i * 4] =		make_float4(rot[0], point.x);
	res[(i * 4) + 1] =  make_float4(rot[1], point.y);
	res[(i * 4) + 2] =  make_float4(rot[2], point.z);
	res[(i * 4) + 3] =  make_float4(0, 0, 0, 1);
}

//static
void CPFToolsGPU::GetRefFrames(vector<Eigen::Affine3f>& dst, vector<Eigen::Vector3f>& p, vector<Eigen::Vector3f>& n)
{
	//Map Eigen vectors to float3 pointers
	vecToPointer3F(vectorsA, p);
	vecToPointer3F(vectorsB, n);

	int threads = 32;
	int blocks = p.size() / threads;
	GetRefFrameGPU<<<blocks, threads>>>(vectorsA, vectorsB, p.size(), RefFrames);

	hipDeviceSynchronize();

	std::vector<Eigen::Affine3f> frames = std::vector<Eigen::Affine3f>(p.size());
	pointerToVecM4F(dst, RefFrames);
}



//TODO
__global__
void DiscretizeCurvatureGPU(float2* dst, float3* n1, float3* n, Matches* matches, int* range, int iteration) 
{
	int i = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (matches[i].matches[iteration].distance > 0.0)
	{
		int id = matches[i].matches[iteration].second;
		dst[i].x += AngleBetweenGPU(n1[i], n[id]);
		dst[i].y++;
	}
}

__global__
void CalculateDiscCurve(int* dst, float2* src)
{
	int i = (blockIdx.x * blockDim.x) + threadIdx.x;
	dst[i] = src[i].x / src[i].y;
}

//static
void CPFToolsGPU::DiscretizeCurvature(vector<uint32_t>& dst, const vector<Eigen::Vector3f>& n1, PointCloud& pc, const Matches* matches, const float range)
{
	vecToPointer3F(vectorsA, n1);
	vecToPointer3F(pcN, pc.normals);
	curvature_pairs = (float2*)malloc(n1.size() * sizeof(float2));
	hipMemcpy(pt_matches, matches, n1.size() * sizeof(Matches), hipMemcpyHostToDevice);
	*int_p = range;
	for (int i = 0; i < n1.size(); i++)
		curvature_pairs[i] = make_float2(0, 0);

	int threads = 64;
	int blocks = pc.size() / threads;
	for (int i = 0; i < 21; i++)
	{
		DiscretizeCurvatureGPU<<<blocks, threads>>>(curvature_pairs, vectorsA, pcN, pt_matches, int_p, i);
	}

	CalculateDiscCurve<<<blocks, threads>>>(discretized_curvatures, curvature_pairs);

	hipDeviceSynchronize();

	pointerToVecI(dst, discretized_curvatures);
}



//TODO
__global__
void DiscretizeCPFGPU(CPFDiscreet* dst, uint32_t* curvatures, Matches* matches, int iteration)
{
	int i = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (matches[i].matches[iteration].distance > 0.0) {
		int id = matches[i].matches[iteration].second;
		int cur1 = curvatures[i];
		int cur2 = curvatures[id];

		float3 pt = ref_frames[i] * pts[id];
	}


}

//TODO
//static
void CPFToolsGPU::DiscretizeCPF(vector<CPFDiscreet>& dst, vector<uint32_t>& curvatures, Matches* matches, int num_matches, vector<Eigen::Vector3f> pts, vector<Eigen::Affine3f> ref_frames)
{
	for (int i = 0; i < num_matches; i++)
	{
		DiscretizeCPFGPU()
	}
}